#include "io.h"
#include "md_kernel.cuh"
#include "vtk_writer.h"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <string>
#include "cell_list.h"

int main(int argc, char** argv) {
    if (argc < 10) {
        std::cerr << "Usage: " << argv[0] << " input.txt dt nsteps sigma epsilon" << std::endl;
        return 1;
    }
    std::string input_file = argv[1];
    float dt = std::stof(argv[2]);
    int nsteps = std::stoi(argv[3]);
    float sigma = std::stof(argv[4]);
    float epsilon = std::stof(argv[5]);
    float box_x = std::stof(argv[6]);
    float box_y = std::stof(argv[7]);
    float box_z = std::stof(argv[8]);
    float rcut  = std::stof(argv[9]);
    float max_radius = 0.0f;
    for (const auto& p : particles) {
        if (p.radius > max_radius) max_radius = p.radius;
    }
    float cell_size = 2.0f * max_radius; 
    int ncell_x = int(box_x / cell_size);
    int ncell_y = int(box_y / cell_size);
    int ncell_z = int(box_z / cell_size);
    CellList clist(ncell_x, ncell_y, ncell_z, cell_size);
    clist.cell_size = cell_size;

    std::vector<Particle> particles;
    if (!read_particles(input_file, particles)) return 1;
    int N = particles.size();

    // Allocate device memory
    Particle* d_particles;
    hipMalloc(&d_particles, N * sizeof(Particle));
    hipMemcpy(d_particles, particles.data(), N * sizeof(Particle), hipMemcpyHostToDevice);

    const char* outdir_env = std::getenv("OUTPUT_DIR");
    std::string outdir = outdir_env ? outdir_env : "output";

    auto t_start = std::chrono::high_resolution_clock::now();

    launch_compute_forces(d_particles, N, sigma, epsilon, box_x, box_y, box_z, rcut);

    for (int step = 0; step < nsteps; ++step) {
        launch_integrate_first_half(d_particles, N, dt, box_x, box_y, box_z);
        hipMemcpy(particles.data(), d_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
        build_cell_list(particles, clist, box_x, box_y, box_z);
        compute_forces_cell_list(particles, clist, sigma, epsilon, box_x, box_y, box_z, rcut);
        hipMemcpy(d_particles, particles.data(), N * sizeof(Particle), hipMemcpyHostToDevice);
        launch_compute_forces(d_particles, N, sigma, epsilon, box_x, box_y, box_z, rcut);
        launch_integrate_second_half(d_particles, N, dt);
        // Output VTK every 100 steps
        if (step % 10 == 0) {
            hipMemcpy(particles.data(), d_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
            for (int i = 0; i < N; ++i) {
                std::cout << "Step " << step << " Particle " << i
                          << ": pos=(" << particles[i].pos.x << "," << particles[i].pos.y << "," << particles[i].pos.z << ")"
                          << " vel=(" << particles[i].vel.x << "," << particles[i].vel.y << "," << particles[i].vel.z << ")\n";
            }
            std::string vtkfile = outdir + "/step_" + std::to_string(step) + ".vtk";
            write_vtk(vtkfile, particles, step);
        }
        
    }

    auto t_end = std::chrono::high_resolution_clock::now();
    double elapsed = std::chrono::duration<double>(t_end - t_start).count();
    std::cout << "Total simulation time: " << elapsed << " s\n";
    std::cout << "Average time per step: " << (elapsed / nsteps) << " s\n";

    hipMemcpy(particles.data(), d_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
    hipFree(d_particles);

    void print_particles(const std::vector<Particle>& particles, int max_print);
    print_particles(particles, 5);
    return 0;
}